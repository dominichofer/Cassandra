#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "Board/Board.h"
#include "DeviceVector.cuh"
#include "HostVector.cuh"
#include ""
#include <cstdint>
#include <omp.h>
#include <numeric>

#include "Board/Position.cpp"
#include "Board/PossibleMoves.cpp"
#include "Board/Field.cpp"
#include "Board/Flips.cpp"

// perft for 0 plies left
__host__ __device__ int64_t perft_0()
{
    return 1;
}

// perft for 1 ply left
__host__ __device__ int64_t perft_1(const Position& pos)
{
    auto moves = PossibleMoves(pos);
    if (moves)
        return moves.size();
    return PossibleMoves(PlayPass(pos)) ? 1 : 0;
}

// perft for 2 plies left
__host__ __device__ int64_t perft_2(const Position& pos)
{
    auto moves = PossibleMoves(pos);
    if (!moves)
        return PossibleMoves(PlayPass(pos)).size();

    int64_t sum = 0;
    for (Field move : moves)
        sum += perft_1(Play(pos, move));
    return sum;
}

__host__ __device__ int64_t perft_3(const Position& pos)
{
    auto moves = PossibleMoves(pos);
    if (!moves)
    {
        Position passed = PlayPass(pos);
        if (PossibleMoves(passed))
            return perft_2(passed);
        return 0;
    }

    int64_t sum = 0;
    for (Field move : moves)
        sum += perft_2(Play(pos, move));
    return sum;
}

__global__ void perft_3(const CudaVector_view<Position> pos, CudaVector_view<int64_t> result)
{
    unsigned int gridSize = blockDim.x * gridDim.x;
    for (unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; i < pos.size(); i += gridSize)
        result[i] = perft_3(pos[i]);
}

__host__ int64_t perft_cuda(const Position& pos, const int depth, const int cuda_depth)
{
    static thread_local int tid = []() { int n; hipGetDeviceCount(&n); int tid = omp_get_thread_num() % n; hipSetDevice(tid); return tid; }();
    static thread_local PinnedVector<Position> positions;
    static thread_local PinnedVector<int64_t> result;
    static thread_local CudaVector<Position> cuda_pos;
    static thread_local CudaVector<int64_t> cuda_result;

    auto gen = Children(pos, depth - cuda_depth, true);
    positions.store(gen.begin(), gen.end());

    cuda_pos.store(positions, asyn);
    cuda_result.resize(positions.size(), asyn);
    assert(cuda_depth == 3);
    perft_3<<<256, 128>>>(cuda_pos, cuda_result);
    result.store(cuda_result, asyn);
    hipDeviceSynchronize();

    return std::accumulate(result.begin(), result.end(), static_cast<int64_t>(0));
}